#include "hip/hip_runtime.h"
/**
 * (C) Copyright 2020, 2021, 2022, 2023, 2024 IBM. All Rights Reserved.
 *
 * Licensed under the MIT license. See LICENSE file in the project root for details.
 */

#include "pwu_kernel_parameter.h"
#include "rpu_pulsed_meta_parameter.h"
#include "rpucuda_custom_device.h"

namespace RPU {

template <typename T> struct UpdateFunctorCustom {

  __device__ __forceinline__ void operator()(
      T &w,
      uint32_t n,
      uint32_t negative,
      const param4_t par_4,
      const param2_t par_2,
      T &par_1,
      const T *global_par,
      const int global_params_count,
      T noise_std_dw,
      hiprandState &local_state) {

    // note that only w and par_1 will be written back when used. Thus it can be a "hidden_weights"
    // type note that we here assume that stoch_value is < 1, or if larger, then it did not hit the
    // bound.
    UNUSED(global_params_count);
    UNUSED(global_par);
    UNUSED(par_1);
    UNUSED(par_2);

    T dw = (negative > 0) ? ((T)par_4.w) : (-(T)par_4.y);
    T wmax = (T)par_4.z;
    T wmin = (T)par_4.x;
    T sigma = noise_std_dw;
    // n is larger 0 in any case
    if (n == 1) {
      if (sigma > (T)0.0) {
        T stoch_value = (T)hiprand_normal(&local_state);
        stoch_value *= sigma;
        w += dw * ((T)1.0 + stoch_value);
      } else {
        w += dw;
      }
    } else {
      if (sigma > (T)0.0) {
        T stoch_value = (T)hiprand_normal(&local_state);
        stoch_value *= sigma;
        w += dw * (T)n * ((T)1.0 + rsqrt((T)n) * stoch_value); // rsqrt(x) = 1/sqrt(x) is faster
      } else {
        w += dw * (T)n;
      }
    }

    // better always check both bounds
    w = (w > wmax) ? wmax : w;
    w = (w < wmin) ? wmin : w;
  }
};


template <typename T> struct UpdateFunctorCustomLargeNoise {

  __device__ __forceinline__ void operator()(
      T &w,
      uint32_t n,
      uint32_t negative,
      const param4_t par_4,
      const param2_t par_2,
      T &par_1,
      const T *global_par,
      const int global_params_count,
      T noise_std_dw,
      hiprandState &local_state) {

    UNUSED(global_params_count);
    UNUSED(global_par);
    UNUSED(par_1);
    UNUSED(par_2);
    // negative > 0 means going up here ...
    // here we assume that noise_std_dw>0 at least
    T wmax = par_4.z;                                   // [2];
    T wmin = par_4.x;                                   //[0];
    float dw = (negative > 0) ? (par_4.w) : (-par_4.y); // [3], [1]
    float sigma = noise_std_dw;

    // n is larger 0 in any case
    if (n == 1) { // short-cut without loop
      float stoch_value = hiprand_normal(&local_state);
      stoch_value *= sigma;
      w += dw * ((float)1.0 + stoch_value);

      w = (w > wmax) ? wmax : w;
      w = (w < wmin) ? wmin : w;

    } else {
      for (int i = 0; i < n; i++) { // need to loop here because noise can be large and hit the
                                    // boundary and retract again because of sign reverse
        float stoch_value = hiprand_normal(&local_state);
        stoch_value *= sigma;
        w += dw * ((float)1.0 + stoch_value);

        w = (w > wmax) ? wmax : w;
        w = (w < wmin) ? wmin : w;
      }
    }
  }
};

#define ARGS(NAME)                                                                                 \
  (this->context_, this->x_size_, this->d_size_, m_batch, nK32, use_bo64, out_trans, up,           \
   getPar().getName() + #NAME)

template <typename T>
pwukpvec_t<T> CustomRPUDeviceCuda<T>::getUpdateKernels(
    int m_batch, int nK32, int use_bo64, bool out_trans, const PulsedUpdateMetaParameter<T> &up) {

  pwukpvec_t<T> v;

  if (getPar().dw_min_std > (T)0.33) { // 3 sigma
    v.push_back(
        RPU::make_unique<
            PWUKernelParameterSingleFunctor<T, UpdateFunctorCustomLargeNoise<T>, 1>>
            ARGS(FunctorLargeNoise));
    v.push_back(
        RPU::make_unique<
            PWUKernelParameterBatchFunctor<T, UpdateFunctorCustomLargeNoise<T>, 1>>
            ARGS(FunctorLargeNoise));
    v.push_back(
        RPU::make_unique<
            PWUKernelParameterBatchSharedFunctor<T, UpdateFunctorCustomLargeNoise<T>, 1>>
            ARGS(FunctorLargeNoise));
    v.push_back(
        RPU::make_unique<PWUKernelParameterBatchSharedWeightOutputFunctor<
            T, UpdateFunctorCustomLargeNoise<T>, 1>> ARGS(FunctorLargeNoise));

  } else {
    // use summing approximation is save in this case
    // Update functor and kernels are in pwu_kernels.h
    v.push_back(
        RPU::make_unique<PWUKernelParameterBatchSharedFunctor<T, UpdateFunctorCustom<T>, 1>>
            ARGS(Functor));
    v.push_back(
        RPU::make_unique<
            PWUKernelParameterBatchSharedWeightOutputFunctor<T, UpdateFunctorCustom<T>, 1>>
            ARGS(Functor));
    v.push_back(
        RPU::make_unique<PWUKernelParameterBatchFunctor<T, UpdateFunctorCustom<T>, 1>> ARGS(
            Functor));

    v.push_back(
        RPU::make_unique<PWUKernelParameterSingleFunctor<T, UpdateFunctorCustom<T>, 1>> ARGS(
            Functor));
    v.push_back(RPU::make_unique<PWUKernelParameterBatchSharedSum<T>> ARGS(Sum));
    v.push_back(RPU::make_unique<PWUKernelParameterBatchSharedSumBoundCheck<T>> ARGS(SumBC));

    v.push_back(RPU::make_unique<PWUKernelParameterBatchSum<T>> ARGS(Sum));
    v.push_back(RPU::make_unique<PWUKernelParameterBatchSumBoundCheck<T>> ARGS(SumBC));
  }

  return v;
}

#undef ARGS

template class CustomRPUDeviceCuda<float>;
#ifdef RPU_USE_DOUBLE
template class CustomRPUDeviceCuda<double>;
#endif
#ifdef RPU_USE_FP16
template class CustomRPUDeviceCuda<half_t>;
#endif

} // namespace RPU
